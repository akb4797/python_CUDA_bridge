#include "hip/hip_runtime.h"

#include "book.h"
#include "cudacode.h"


__global__ void findQueryMatch_GPU(char* inpArray, char* queiryArray, int* retVal, int sizeInput)
{
	//printf("blockIdx.x:%d * blockDim.x:%d + threadIdx.x:%d \n", blockIdx.x, blockDim.x, threadIdx.x);

    int itr = (blockIdx.x* blockDim.x + threadIdx.x);
   // printf("\nitr : %d", itr);

    if((itr < sizeInput) && (itr%2 ==0))
    {     
        int xorVal = (inpArray[itr] ^ queiryArray[itr%2] || inpArray[itr+1] ^ queiryArray[(itr+1)%2]);
        //printf("\nXor param : %d, %d", (valOne),valTwo);
    /*
        printf("\nInp : %c, %c", inpArray[itr],inpArray[itr+1]);
        printf("\nQuer : %c, %c", queiryArray[itr%2], queiryArray[(itr%2)+1]);
        printf("\nXor : %d", xorVal);
    */

        if(xorVal == 0)
        {
            *retVal = 1;
            //printf("\nretVal : %d", *retVal);
        }


    }
    
 

}


cudaInterface::cudaInterface()
{

	hipDeviceProp_t	prop;
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
	
	for (int i=0; i< count; i++) {
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		std::cout << "GPU Device name : " << prop.name << std::endl;
		std::cout << "totalGlobalMem : " << prop.totalGlobalMem << std::endl;
		std::cout << "sharedMemPerBlock : " << prop.sharedMemPerBlock << std::endl;

		printf( "Max thread dimensions:	(%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
		printf( "Max grid dimensions:	(%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );

	}

    queiryBlockSize = 2;
    inputBlockSize = queiryBlockSize;

    MAX_GT_NUM = 4;
    MAX_GT_SIZE = 2;

    inputPtr = new char[MAX_GT_NUM*MAX_GT_SIZE];
    queiryPtr = new char[MAX_GT_SIZE];

    inptTailPtr = inputPtr;

	//hipSetDevice(0);
}

void cudaInterface::checkMatch()
{
    printf("\nCUDA driver call..");
    int matchFoundFlag= 0;

    queiryBlockSize = 2;
    inputBlockSize = 4;

    char inputArray[] = {'a','b','c','d'};
    char queiryArray[] = {'c','d'};



	char* d_input;
	char* d_query;
    int* ptr;

	hipMalloc((void **)&d_input,inputBlockSize*sizeof(char)) ;
	hipMalloc((void**)&d_query,queiryBlockSize*sizeof(char)) ;
    hipMalloc((void**)&ptr, sizeof(int));

	hipMemcpy(d_input, inputArray, inputBlockSize*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_query, queiryArray, queiryBlockSize*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(ptr, &matchFoundFlag, sizeof(int), hipMemcpyHostToDevice);


    findQueryMatch_GPU <<<1,1000 >>> (d_input, d_query, ptr, inputBlockSize);

    hipMemcpy(&matchFoundFlag, ptr, sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    if(matchFoundFlag)
    {
        printf(" : Match");
    }
    else
    {
        printf(" : No Match");
    }
	//hipMemcpy(outArray, d_output, 5*sizeof(int), hipMemcpyDeviceToHost);
	

	hipFree(d_input);
	hipFree(d_query);
    hipFree(ptr);

}

int cudaInterface::searchFor(char* quieryString)
{
    /*
    Returns : 
    1 : Match found
    0 : NO match found
    */
    int matchFoundFlag= 0;
    int retVal = 0;

    queiryBlockSize = MAX_GT_SIZE;
    inputBlockSize = MAX_GT_NUM*MAX_GT_SIZE;

    
    

    //Skip check if cold start
    if(inptTailPtr == inputPtr)
    {
        printf("\nInitiating Memory\n");

        inptTailPtr += MAX_GT_SIZE;
        strcpy(inputPtr, quieryString);
        retVal = 1;
    }
    else
    {
        //printf("\nScanning Memory");
        strcpy(queiryPtr, quieryString);
        inptTailPtr += MAX_GT_SIZE;


        char* d_input;
        char* d_query;
        int* ptr;
    
        hipMalloc((void **)&d_input,inputBlockSize*sizeof(char)) ;
        hipMalloc((void**)&d_query,queiryBlockSize*sizeof(char)) ;
        hipMalloc((void**)&ptr, sizeof(int));
    
        hipMemcpy(d_input, inputPtr, inputBlockSize*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_query, queiryPtr, queiryBlockSize*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(ptr, &matchFoundFlag, sizeof(int), hipMemcpyHostToDevice);
    
    
        findQueryMatch_GPU <<<1,1000 >>> (d_input, d_query, ptr, inputBlockSize);
    
        hipMemcpy(&matchFoundFlag, ptr, sizeof(int), hipMemcpyDeviceToHost);
    
        hipDeviceSynchronize();
    
        if(matchFoundFlag)
        {
            printf(" : Match\n");
            retVal = 1;
        }
        else
        {
            printf(" : No Match\n");
            strcpy(inptTailPtr, quieryString);

        }
        //hipMemcpy(outArray, d_output, 5*sizeof(int), hipMemcpyDeviceToHost);
        
    
        hipFree(d_input);
        hipFree(d_query);
        hipFree(ptr);
    
    }

    return retVal;

 
}

void cudaInterface::testBenchDriver()
{
#if 1
    std::cout << "TEST BENCH" << std::endl;
    std::string testString = "ab";

    char testArrayOne[] =  {'a','b'};
    char testArrayTwo[] =  {'d','e'};
    char testArrayFour[] =  {'x','y'};

    int retVal = this->searchFor(testArrayOne);
    retVal = this->searchFor(testArrayTwo);
    retVal = this->searchFor(testArrayOne);
    retVal = this->searchFor(testArrayTwo);
    retVal = this->searchFor(testArrayFour);
    std::cout << "EXPECTED OUTPUT : NM, M, M, NM"<< std::endl;
#endif
}